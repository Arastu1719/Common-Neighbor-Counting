#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "Graph.h"
#include "Algorithm3.cuh"
#include "Algorithm6.cuh"
int Threshold = 2;

int main() {

    Graph g;
    g.readGraph();

    int* dev_offset, * dev_dst, * dev_cnt, * dev_Threshold, * dev_n, * dev_m;
    bool* B;
    hipMalloc((void**)&dev_offset, (g.n + 1) * sizeof(int));
    hipMalloc((void**)&dev_dst, 2 * g.m * sizeof(int));
    hipMalloc((void**)&dev_cnt, 2 * g.m * sizeof(int));
    hipMalloc((void**)&dev_Threshold, sizeof(int));
    hipMalloc((void**)&dev_n, sizeof(int));
    hipMalloc((void**)&dev_m, sizeof(int));
    hipMalloc((void**)&B, g.n * sizeof(bool));

    for (int i = 0; i <= g.n; ++i) {
        hipMemcpy(&dev_offset[i], &g.offset[i], sizeof(int), hipMemcpyHostToDevice);
        if (i < g.n) {
            bool temp = false;
            hipMemcpy(&B[i], &temp, sizeof(bool), hipMemcpyHostToDevice);
        }
    }

    for (int i = 0; i < 2 * g.m; ++i) {
        hipMemcpy(&dev_dst[i], &g.dst[i], sizeof(int), hipMemcpyHostToDevice);
    }
    hipMemcpy(dev_Threshold, &Threshold, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_n, &g.n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_m, &g.m, sizeof(int), hipMemcpyHostToDevice);

    int p = sqrt(g.n);
    if (p * p < g.n) ++p;
    IntersectBMP << <p, 1, g.n * sizeof(bool)>> > (dev_n, dev_m, dev_offset, dev_dst, dev_cnt);
    hipDeviceSynchronize();
    int* c = (int*)malloc(2 * g.m * sizeof(int));
    hipMemcpy(c, dev_cnt, 2 * g.m * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 2 * g.m; ++i)
        printf("%d ", c[i]);

    return 0;
}